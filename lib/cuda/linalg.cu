#include "hip/hip_runtime.h"
// -*- C++ -*-
// -*- coding: utf-8 -*-
//
// Lijun Zhu
// california institute of technology
// (c) 2016-2019  all rights reserved
//

// my declaration
#include "linalg.h"
// cuda utilities
#include "cudalib.h"
#include <iostream>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

namespace linalg_kernels {

    template<typename T, const size_t Nthreads>
    __global__ void _determinant(const T * const g_idata, T * const g_odata, const size_t n);

    template<typename T, const size_t Nthreads>
    __global__ void _logdet(const T * const g_idata, T * const g_odata, const size_t n);

}

// put specialization inside namespace for gcc<7 compatibility
namespace cudalib {
    namespace linalg {

// matrix inverse with LU
// double specialization
template <>
int
inverse_lu_cublas<double>(hipblasHandle_t handle, double * const matrix, const size_t n, hipStream_t stream)
{

    hipblasSetStream(handle, stream);

    int batchSize = 1;

    int *P, *INFO;

    cudaSafeCall(hipMalloc((void **)&P,n * batchSize * sizeof(int)));
    cudaSafeCall(hipMalloc((void **)&INFO,batchSize * sizeof(int)));

    int lda = n;

    double *A[] = { matrix };
    double** A_d;
    cudaSafeCall(hipMalloc<double*>(&A_d,sizeof(A)));
    cudaSafeCall(hipMemcpy(A_d,A,sizeof(A),hipMemcpyHostToDevice));

    cublasSafeCall(hipblasDgetrfBatched(handle,n,A_d,lda,P,INFO,batchSize));

    int INFOh = 0;
    cudaSafeCall(hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh == n)
    {
        fprintf(stderr, "Factorization Failed: Matrix is singular\n");
        hipDeviceReset();
    }

    cublasSafeCall(hipblasDgetriBatched(handle,n,A_d,lda,P,A_d,lda,INFO,batchSize));

    cudaSafeCall(hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh != 0)
    {
        fprintf(stderr, "Inversion Failed: Matrix is singular\n");
        hipDeviceReset();
    }
    hipFree(P), hipFree(INFO), hipFree(A_d);
    return INFOh;
}

// float sepcialization
template <>
int
inverse_lu_cublas<float>(hipblasHandle_t handle, float * const matrix, const size_t n, hipStream_t stream)
{
    // set stream
    hipblasSetStream(handle, stream);
    // one matrix
    int batchSize = 1;

    int *P, *INFO;

    cudaSafeCall(hipMalloc((void **)&P, n*batchSize* sizeof(int)));
    cudaSafeCall(hipMalloc((void **)&INFO, batchSize*sizeof(int)));

    int lda = n;

    float *A[] = { matrix };
    float** A_d;
    cudaSafeCall(hipMalloc<float*>(&A_d,sizeof(A)));
    cudaSafeCall(hipMemcpy(A_d,A,sizeof(A),hipMemcpyHostToDevice));

    cublasSafeCall(hipblasSgetrfBatched(handle,n,A_d,lda,P,INFO,batchSize));

    int INFOh = 0;
    cudaSafeCall(hipMemcpy((void **)&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh == n)
    {
        fprintf(stderr, "Factorization Failed: Matrix is singular\n");
        hipDeviceReset();
    }

    cublasSafeCall(hipblasSgetriBatched(handle,n,A_d,lda,P,A_d,lda,INFO,batchSize));
    cudaSafeCall(hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh != 0)
    {
        fprintf(stderr, "Inversion Failed: Matrix is singular\n");
        hipDeviceReset();
    }
    hipFree(P), hipFree(INFO), hipFree(A_d);
    return INFOh;
}

// LU factorization
// double specialization
template<>
int lu<double>(hipsolverHandle_t solver_handle,
                double * const matrix, const size_t lda, const size_t m, const size_t n,
                hipStream_t stream)
{
    hipsolverSetStream(solver_handle, stream);

    int work_size = 0;
    int *devInfo;
    cudaSafeCall(hipMalloc((void **)&devInfo, sizeof(int)));

    int *devIpiv; //pivot indices
    cudaSafeCall(hipMalloc((void **)&devIpiv, min(m,n)*sizeof(int)));

    // get work_size
    // note that cusolver uses column-major, or m is leading dimension
    cusolverSafeCall(hipsolverDnDgetrf_bufferSize(solver_handle,  m, n, matrix, lda, &work_size));

    // allocate work
    double *work;
    cudaSafeCall(hipMalloc((void **)&work, work_size * sizeof(double)));

    // call LU
    cusolverSafeCall(hipsolverDnDgetrf(solver_handle, m, n, matrix, lda, work, devIpiv, devInfo));

    // check error
    int info;
    cudaSafeCall(hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));

    if(info!=0) {
        if(info <0)
            fprintf(stderr, "LU factorization error:  the %d-th parameter is wrong (not counting the handle)\n", -info);
        else
            fprintf(stderr, "LU factorization error:  U(%d, %d) =0 \n", info, info);
    }

    cudaSafeCall(hipFree(work));
    cudaSafeCall(hipFree(devInfo));
    return info;
}

// float specialization
template<>
int lu<float>(hipsolverHandle_t solver_handle,
                float * const matrix, const size_t lda, const size_t m, const size_t n,
                hipStream_t stream)
{

    hipsolverSetStream(solver_handle, stream);

    int work_size = 0;
    int *devInfo;
    cudaSafeCall(hipMalloc((void **)&devInfo, sizeof(int)));

    int *devIpiv; //pivot indices
    cudaSafeCall(hipMalloc((void **)&devIpiv, min(m,n)*sizeof(int)));

    // get work_size
    // note that cusolver uses column-major, or m is leading dimension
    cusolverSafeCall(hipsolverDnSgetrf_bufferSize(solver_handle,  m, n, matrix, lda, &work_size));

    //
    float *work;
    cudaSafeCall(hipMalloc((void **)&work, work_size * sizeof(float)));
    cusolverSafeCall(hipsolverDnSgetrf(solver_handle, m, n, matrix, lda, work, devIpiv, devInfo));

    // check error
    int info;
    cudaSafeCall(hipMemcpy((void **)&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));

    if(info!=0) {
        if(info <0)
            fprintf(stderr, "LU factorization error:  the %d-th parameter is wrong (not counting the handle)\n", -info);
        else
            fprintf(stderr, "LU factorization error:  U(%d, %d) =0 \n", info, info);
        hipDeviceReset();
    }
    // free working space
    cudaSafeCall(hipFree(work));
    cudaSafeCall(hipFree(devInfo));
    return info;
}

// inverse by Cholesky for symmetric matrices
// double specialization
template <>
int
inverse_cholesky<double>(hipsolverHandle_t solver_handle,
        double * const matrix, hipblasFillMode_t uplo, const size_t n,
        hipStream_t stream)
{
    hipsolverSetStream(solver_handle, stream);

    int work_size = 0;
    int *devInfo;
    cudaSafeCall(hipMalloc((void **)&devInfo, sizeof(int)));

    // get work_size
    // note that cusolver uses column-major
    cusolverSafeCall(hipsolverDnDpotrf_bufferSize(solver_handle, uplo, n, matrix, n, &work_size));

    // --- CUDA POTRF execution
    double *work;
    cudaSafeCall(hipMalloc((void **)&work, work_size * sizeof(double)));
    cusolverSafeCall(hipsolverDnDpotrf(solver_handle, uplo, n, matrix, n, work, work_size, devInfo));
    // check error
    int info;
    cudaSafeCall(hipMemcpyAsync(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost, stream));

    if(info!=0) {
        if(info <0)
            fprintf(stderr, "Chelosky factorization error:  the %d-th parameter is wrong (not counting the handle)\n", -info);
        else
            fprintf(stderr, "Chelosky factorization error:  the leading minor of order %d is not positive definite\n", info);
        hipDeviceReset();
    }

    // to be safe, allocate a new work for hipsolverDnDpotri
    int info2=0;
    int * devInfo2;
    cudaSafeCall(hipMalloc((void **)&devInfo2, sizeof(int)));
    int work_size2;
    cusolverSafeCall(hipsolverDnDpotri_bufferSize(solver_handle, uplo, n, matrix, n, &work_size2));
    double * work2;
    cudaSafeCall(hipMalloc((void **)&work2, work_size2 * sizeof(double)));

    // call inverse
    cusolverSafeCall(hipsolverDnDpotri(solver_handle, uplo, n, matrix, n, work2, work_size2, devInfo2));
    // check error
    cudaSafeCall(hipMemcpyAsync(&info2, devInfo2, sizeof(int), hipMemcpyDeviceToHost, stream));

    if(info2!=0) {
        if(info <0)
            fprintf(stderr, "Chelosky inverse error:  the %d-th parameter is wrong (not counting the handle)\n", -info);
        else
            fprintf(stderr, "Chelosky inverse error:  the leading minor of order %d is not positive definite\n", info);
        hipDeviceReset();
    }

    cudaSafeCall(hipFree(work));
    cudaSafeCall(hipFree(work2));
    cudaSafeCall(hipFree(devInfo));
    return info;
}

// inverse by Cholesky (symmetric matrix)
template <>
int inverse_cholesky<float>(hipsolverHandle_t solver_handle,
    float * const matrix, hipblasFillMode_t uplo, const size_t n, hipStream_t stream)
{
    hipsolverSetStream(solver_handle, stream);

    int work_size = 0;
    int *devInfo;
    cudaSafeCall(hipMalloc((void **)&devInfo, sizeof(int)));

    // get work_size
    // note that cusolver uses column-major
    cusolverSafeCall(hipsolverDnSpotrf_bufferSize(solver_handle, uplo, n, matrix, n, &work_size));

    // --- CUDA POTRF execution
    float *work;
    cudaSafeCall(hipMalloc((void **)&work, work_size * sizeof(float)));
    cusolverSafeCall(hipsolverDnSpotrf(solver_handle, uplo, n, matrix, n, work, work_size, devInfo));
    // check error
    int info;
    cudaSafeCall(hipMemcpyAsync(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost, stream));

    if(info!=0) {
        if(info <0)
            fprintf(stderr, "Chelosky factorization error:  the %d-th parameter is wrong (not counting the handle)\n", -info);
        else
            fprintf(stderr, "Chelosky factorization error:  the leading minor of order %d is not positive definite\n", info);
        hipDeviceReset();
    }

    // to be safe, allocate a new work for hipsolverDnDpotri
    int work_size2;
    cusolverSafeCall(hipsolverDnSpotri_bufferSize(solver_handle, uplo, n, matrix, n, &work_size2));
    float * work2;
    cudaSafeCall(hipMalloc((void **)&work2, work_size2 * sizeof(float)));

    // call inverse
    cusolverSafeCall(hipsolverDnSpotri(solver_handle, uplo, n, matrix, n, work2, work_size2, devInfo));
    // check error
    cudaSafeCall(hipMemcpyAsync(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost, stream));

    if(info!=0) {
        if(info <0)
            fprintf(stderr, "Chelosky inverse error:  the %d-th parameter is wrong (not counting the handle)\n", -info);
        else
            fprintf(stderr, "Chelosky inverse error:  the leading minor of order %d is not positive definite\n", info);
        hipDeviceReset();
    }

    cudaSafeCall(hipFree(work));
    cudaSafeCall(hipFree(work2));
    cudaSafeCall(hipFree(devInfo));
    //std::cout << "inverse cholesky" << work_size << " " << work_size2 << "\n";
    return info;
}

// inverse by LU with cusolver
// double specialization
template<>
int
inverse_lu_cusolver<double>(hipsolverHandle_t solver_handle,
    double * const matrix, const size_t n, hipStream_t stream)
{
    hipsolverSetStream(solver_handle, stream);

    int work_size = 0;
    int *devInfo;
    cudaSafeCall(hipMalloc((void **)&devInfo, sizeof(int)));

    int *devIpiv; //pivot indices
    cudaSafeCall(hipMalloc((void **)&devIpiv, n*sizeof(int)));

    // get work_size
    // note that cusolver uses column-major, or m is leading dimension
    cusolverSafeCall(hipsolverDnDgetrf_bufferSize(solver_handle,  n, n, matrix, n, &work_size));

    // allocate working space
    double *work;
    cudaSafeCall(hipMalloc((void **)&work, work_size * sizeof(double)));
    // LU decompose P*A = L*U
    cusolverSafeCall(hipsolverDnDgetrf(solver_handle, n, n, matrix, n, work, devIpiv, devInfo));

    // check error
    int info;
    cudaSafeCall(hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));

    if(info!=0) {
        if(info <0)
            fprintf(stderr, "LU factorization error:  the %d-th parameter is wrong (not counting the handle)\n", -info);
        else
            fprintf(stderr, "LU factorization error:  U(%d, %d) =0 \n", info, info);
    }
    // allocate an identity matrix
    double * dB;
    cudaSafeCall(hipMalloc((void **)&dB, n*n*sizeof(double)));
    cudalib::matrix::identity<double>(dB, n, stream);

    // use linear equation solver A x = B; x=A^{-1} if B=I
    cusolverSafeCall(hipsolverDnDgetrs(solver_handle, HIPBLAS_OP_N, n, n,
            matrix, n, devIpiv, dB, n, devInfo));
    // check error
    cudaSafeCall(hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    if(info!=0) {
        if(info <0)
            fprintf(stderr, "LU solver error:  the %d-th parameter is wrong (not counting the handle)\n", -info);
        hipDeviceReset();
    }
    // copy solution back to A
    cudaSafeCall(hipMemcpy(matrix, dB, n*n*sizeof(double), hipMemcpyDeviceToDevice));
    // free working data
    cudaSafeCall(hipFree(work));
    cudaSafeCall(hipFree(devInfo));
    cudaSafeCall(hipFree(dB));
    return info;
}

// float specialization
template<>
int
inverse_lu_cusolver<float>(hipsolverHandle_t solver_handle, float * const matrix, const size_t n, hipStream_t stream)
{
    hipsolverSetStream(solver_handle, stream);

    int work_size = 0;
    int *devInfo;
    cudaSafeCall(hipMalloc((void **)&devInfo, sizeof(int)));

    int *devIpiv; //pivot indices
    cudaSafeCall(hipMalloc((void **)&devIpiv, n*sizeof(int)));

    // get work_size
    // note that cusolver uses column-major, or m is leading dimension
    cusolverSafeCall(hipsolverDnSgetrf_bufferSize(solver_handle,  n, n, matrix, n, &work_size));

    // allocate working space
    float *work;
    cudaSafeCall(hipMalloc((void **)&work, work_size * sizeof(float)));
    // LU decompose P*A = L*U
    cusolverSafeCall(hipsolverDnSgetrf(solver_handle, n, n, matrix, n, work, devIpiv, devInfo));

    // check error
    int info;
    cudaSafeCall(hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));

    if(info!=0) {
        if(info <0)
            fprintf(stderr, "LU factorization error:  the %d-th parameter is wrong (not counting the handle)\n", -info);
        else
            fprintf(stderr, "LU factorization error:  U(%d, %d) =0 \n", info, info);
    }
    // allocate an identity matrix
    float * dB;
    cudaSafeCall(hipMalloc((void **)&dB, n*n*sizeof(float)));
    cudalib::matrix::identity<float>(dB, n, stream);

    // use linear equation solver A x = B; x=A^{-1} if B=I
    cusolverSafeCall(hipsolverDnSgetrs(solver_handle, HIPBLAS_OP_N, n, n,
            matrix, n, devIpiv, dB, n, devInfo));
    // check error
    cudaSafeCall(hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    if(info!=0) {
        if(info <0)
            fprintf(stderr, "LU solver error:  the %d-th parameter is wrong (not counting the handle)\n", -info);
        hipDeviceReset();
    }
    // copy solution back to A
    cudaSafeCall(hipMemcpy(matrix, dB, n*n*sizeof(float), hipMemcpyDeviceToDevice));
    // free working data
    cudaSafeCall(hipFree(work));
    cudaSafeCall(hipFree(devInfo));
    cudaSafeCall(hipFree(dB));
    return info;
}

// Cholesky factorization double specialization
template<>
int
cholesky<double>(hipsolverHandle_t solver_handle,
    double * const matrix, hipblasFillMode_t uplo, const size_t n, hipStream_t stream)
{
    hipsolverSetStream(solver_handle, stream);

    int work_size = 0;
    int *devInfo;
    cudaSafeCall(hipMalloc((void **)&devInfo, sizeof(int)));

    // get work_size
    cusolverSafeCall(hipsolverDnDpotrf_bufferSize(solver_handle, uplo, n, matrix, n, &work_size));

    // --- CUDA POTRF execution
    double *work;
    cudaSafeCall(hipMalloc((void **)&work, work_size * sizeof(double)));
    cusolverSafeCall(hipsolverDnDpotrf(solver_handle, uplo, n, matrix, n, work, work_size, devInfo));
    // check error
    int info;
    cudaSafeCall(hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));

    if(info!=0) {
        if(info <0)
            fprintf(stderr, "Chelosky factorization error:  the %d-th parameter is wrong (not counting the handle)\n", -info);
        else
            fprintf(stderr, "Chelosky factorization error:  the leading minor of order %d is not positive definite\n", info);
        hipDeviceReset();
    }
    cudaSafeCall(hipFree(work));
    cudaSafeCall(hipFree(devInfo));
    return info;
}

// Cholesky float specialization
template<>
int
cholesky<float>(hipsolverHandle_t solver_handle,
    float * const matrix, hipblasFillMode_t uplo, const size_t n, hipStream_t stream)
{
    hipsolverSetStream(solver_handle, stream);

    int work_size = 0;
    int *devInfo;
    cudaSafeCall(hipMalloc((void **)&devInfo, sizeof(int)));

    // --- CUDA CHOLESKY initialization, get work_size
    // note that cusolver uses column-major
    cusolverSafeCall(hipsolverDnSpotrf_bufferSize(solver_handle, uplo, n, matrix, n, &work_size));

    // --- CUDA POTRF execution
    float *work;
    cudaSafeCall(hipMalloc((void **)&work, work_size * sizeof(float)));
    cusolverSafeCall(hipsolverDnSpotrf(solver_handle, uplo, n, matrix, n, work, work_size, devInfo));
    // check error
    int info;
    cudaSafeCall(hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));

    if(info!=0) {
        if(info <0)
            fprintf(stderr, "Chelosky factorization error:  the %d-th parameter is wrong (not counting the handle)\n", -info);
        else
            fprintf(stderr, "Chelosky factorization error:  the leading minor of order %d is not positive definite\n", info);
        hipDeviceReset();
    }
    cudaSafeCall(hipFree(work));
    cudaSafeCall(hipFree(devInfo));
    return info;
}

//
template<typename T>
T
determinant_triangular(const T * const mat, const size_t n, hipStream_t stream)
{
    int nthreads = NTHREADS;
    int nblocks = IDIVUP(n, 2*nthreads);
    dim3 blockSize (nthreads, 1, 1);
    dim3 gridSize  (nblocks, 1, 1);

    // create a work vector if n > nthreads
    T *hprod = (T *) malloc(nblocks * sizeof(T));
    T *dprod = NULL;
    cudaSafeCall(hipMalloc((void **) &dprod, nblocks * sizeof(T)));

    // reduce product in each block by gpu
    linalg_kernels::_determinant<T, NTHREADS><<<gridSize, blockSize, 0, stream>>>
        (mat, dprod, n);
    cudaCheckError("linalg_kernels::determinant error");

    cudaSafeCall(hipMemcpy(hprod, dprod, nblocks * sizeof(T),hipMemcpyDeviceToHost));
    cudaSafeCall(hipFree(dprod));

    T product = (T)1.0f;

    // reduce product for blocks
    for (int i = 0; i < nblocks; i++)
    {
        product *= hprod[i];
    }
    free(hprod);
    return product;
}

// explicit instantiation for shared library
template float determinant_triangular<float>(const float * const, const size_t, hipStream_t);
template double determinant_triangular<double>(const double * const, const size_t, hipStream_t);
template int determinant_triangular<int>(const int * const, const size_t, hipStream_t);

//
template<typename T>
T
logdet_triangular(const T * const mat, const size_t n, hipStream_t stream)
{
    int nthreads = NTHREADS;
    int nblocks = IDIVUP(n, 2*nthreads);
    dim3 blockSize (nthreads, 1, 1);
    dim3 gridSize  (nblocks, 1, 1);

    // create a work vector if n > nthreads
    T *hprod = (T *) malloc(nblocks * sizeof(T));
    T *dprod = NULL;
    cudaSafeCall(hipMalloc((void **) &dprod, nblocks * sizeof(T)));

    // reduce product in each block by gpu
    linalg_kernels::_logdet<T, NTHREADS><<<gridSize, blockSize, 0, stream>>>
        (mat, dprod, n);
    cudaCheckError("linalg_kernels::determinant error");

    cudaSafeCall(hipMemcpy(hprod, dprod, nblocks * sizeof(T),hipMemcpyDeviceToHost));
    cudaSafeCall(hipFree(dprod));

    T product = 0.0f;

    // reduce product for blocks
    for (int i = 0; i < nblocks; i++)
    {
        product += hprod[i];
    }
    free(hprod);
    return product;
}

// explicit instantiation for shared library
template float logdet_triangular<float>(const float * const, const size_t, hipStream_t);
template double logdet_triangular<double>(const double * const, const size_t, hipStream_t);

template<typename T>
T determinant_cusolver(hipsolverHandle_t handle, T * const mat, const size_t n, hipStream_t stream)
{
    // Cholesky decomposition
    int info = cudalib::linalg::cholesky<T>(handle, mat, HIPBLAS_FILL_MODE_LOWER, n, stream);
    // Use diagonal part
    T product = cudalib::linalg::determinant_triangular<T>(mat, n, stream);
    return product*product;
}

// explicit instantiation for shared library
template float determinant_cusolver<float>(hipsolverHandle_t, float * const, const size_t, hipStream_t);
template double determinant_cusolver<double>(hipsolverHandle_t, double * const, const size_t, hipStream_t);


    } // of namespace linalg
} // of ns cudalib


// the product of diagonal elements with shfl reduction
// adapted from NVIDIA_CUDA_SAMPLES reduce5

namespace linalg_kernels {
template<typename T, const size_t blockSize>
__global__ void
_determinant(const T * const g_idata, T * const g_odata, const size_t n)
{
       // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    __shared__  T sdata[blockSize];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;

    T localProduct = (i < n) ? g_idata[IDXDIAG(i,n)] : 1;

    if (i + blockSize < n)
        localProduct *= g_idata[IDXDIAG(i+blockSize, n)];

    sdata[tid] = localProduct;
    cg::sync(cta);

    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = localProduct = localProduct * sdata[tid + 256];
    }

    cg::sync(cta);

    if ((blockSize >= 256) &&(tid < 128))
    {
        sdata[tid] = localProduct = localProduct * sdata[tid + 128];
    }

    cg::sync(cta);

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = localProduct = localProduct * sdata[tid +  64];
    }

    cg::sync(cta);

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    if (cta.thread_rank() < 32)
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) localProduct *= sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = tile32.size()/2; offset > 0; offset /= 2)
        {
             localProduct *= tile32.shfl_down(localProduct, offset);
        }
    }

    // write result for this block to global mem
    if (cta.thread_rank() == 0) g_odata[blockIdx.x] = localProduct;
}


template<typename T, const size_t blockSize>
__global__ void
_logdet(const T * const g_idata, T * const g_odata, const size_t n)
{
       // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    __shared__  T sdata[blockSize];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;

    T localSum = (i < n) ? log(g_idata[IDXDIAG(i,n)]) : 0;

    if (i + blockSize < n)
        localSum += log(g_idata[IDXDIAG(i+blockSize, n)]);

    sdata[tid] = localSum;
    cg::sync(cta);

    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = localSum = localSum + sdata[tid + 256];
    }

    cg::sync(cta);

    if ((blockSize >= 256) &&(tid < 128))
    {
        sdata[tid] = localSum = localSum + sdata[tid + 128];
    }

    cg::sync(cta);

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = localSum = localSum + sdata[tid +  64];
    }

    cg::sync(cta);

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    if (cta.thread_rank() < 32)
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) localSum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = tile32.size()/2; offset > 0; offset /= 2)
        {
             localSum += tile32.shfl_down(localSum, offset);
        }
    }

    // write result for this block to global mem
    if (cta.thread_rank() == 0) g_odata[blockIdx.x] = localSum;
}

} // ns linalg_kernels

// end of file
